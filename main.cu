#include "hip/hip_runtime.h"
#include <stdio.h>
#include "pgmProcess.h"
#include "pgmUtility.h"
#include "timing.h"
int main(int argc, char *argv[]){

	double now, then, cost;
    ArgOption opt;
	FILE *fp = NULL, *out = NULL;
	char ** header = (char**) malloc( sizeof(char *) * rowsInHeader);

    int i;
    int * pixels = NULL;

    for(i = 0; i < 4; i++){
        header[i] = (char *) malloc (sizeof(char) * maxSizeHeadRow);
    }

    int numRows, numCols;

    int p1y = 0;
    int p1x = 0;
    int p2y = 0;
    int p2x = 0;

    int m, n, l, x, ch;
    int edgeWidth, circleCenterRow, circleCenterCol, radius;
    char originalImageName[100], newImageFileName[100];

    opt = parseOpt(argc, argv);

    //block of if's for parsing the input for each specific shape type. Once parsed the content of the variables will be changed 
    if (opt == OPT_CIRCLE)
        parseArgsCircle(argv, &circleCenterRow, &circleCenterCol, &radius, originalImageName, newImageFileName);

    if (opt == OPT_EDGE)
        parseArgsEdge(argv, &edgeWidth, originalImageName, newImageFileName);

    if (opt == OPT_LINE)
        parseArgsLine(argv, &p1y, &p1x, &p2y, &p2x, originalImageName, newImageFileName);

    if (opt != OPT_NULL){
        fp = fopen(originalImageName, "r");
        out = fopen(newImageFileName, "w");
    }

    if(fp == NULL || out == NULL || opt == OPT_NULL){
        if(fp != NULL) fclose(fp);
        if(out != NULL) fclose(out);
        usage();
        return 1;
    }

    pixels = pgmRead(header, &numRows, &numCols, fp);

    //GPU METHOD SET_UP
    
    //input data 1d array and 2d array 
    int * d_pixels;
    char ** d_header;

    //output 1D array
    int *o_pixels;


    //number of bytes for the two variables from above. 
    size_t bytes = (sizeof(int) * (numRows * numCols));
    size_t hbytes = (sizeof(char) * maxSizeHeadRow);

    hipMalloc(&d_header, hbytes);
    hipMalloc(&d_pixels, bytes);
    hipMalloc(&o_pixels, bytes);

    //copying host to the device. Pixels being copied to d_pixels. Same for host.
	hipMemcpy(d_pixels, pixels, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_header, header, hbytes, hipMemcpyHostToDevice);
    hipMemcpy(o_pixels, pixels, bytes, hipMemcpyHostToDevice);
	
    //not sure what to do for grid size or n so we're gonna do 100000 like vecAdd example
	int n1 = 100000; 
    int blockSize = 1024; 
    int gridSize;

    //number of threads in a block
	gridSize = (int)ceil((float)n1/blockSize);
    //experimental
    // gridSize = (int)ceil((float) numRows/blockSize);
    //END OF SETUP
    
    //The actuall logic methods that will help create the different shapes on the images.  
    if (opt == OPT_CIRCLE){

    }
    if (opt == OPT_EDGE) {
        //declare device memories needed for edge
        //pgmDrawEdge(pixels, numRows, numCols, edgeWidth, header);
        //drawEdgeCUDA<<<gridSize, blockSize>>>(d_pixels, numRows, numCols, edgeWidth, d_header);   
        //input pixels,inputheader, output pixels, numRows, numC 
        drawEdgeCUDA<<<gridSize, blockSize>>>(d_pixels,d_header,o_pixels,numRows,numCols,edgeWidth);
	}
    if (opt == OPT_LINE){
        pgmDrawLine(pixels, numRows, numCols, header, p1y, p1x, p2y, p2x);
    }
    
    //cuda memcpy back to host
	hipMemcpy(pixels, o_pixels, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(header, d_header, hbytes, hipMemcpyDeviceToHost);

    //timing 
    now = currentTime();
	cost = now - then;	
    printf("Code execution time: %lf\n", cost);

    //once we've done our echanges we are going to pass our one d array and print it out as a 2D-array 
    pgmWrite(header, pixels, numRows, numCols, out );

    //free cuda memory
	hipFree(d_pixels);
	hipFree(d_header);
    hipFree(d_header);

    i = 0;
    for(;i < rowsInHeader; i++)
        free(header[i]);
    free(header);
    if(out != NULL)
        fclose(out);
    if(fp != NULL)
        fclose(fp);

	return 0;
}